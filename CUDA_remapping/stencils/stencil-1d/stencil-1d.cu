#include "hip/hip_runtime.h"
#include<stdio.h>
#include "../../../utilities/remapping.h"
#include "../../../utilities/remapping_mode.h"

// define vector length, stencil radius, 
#define N (GRIDSIZE * BLOCKSIZE * DATA_PER_THREAD)
#define DATA_PER_THREAD 3000 * 2 // 3000 is can fill up the 20MB cache
#define RADIUS 2000
#define GRIDSIZE 13
#define BLOCKSIZE 128

// -------------------------------------------------------
// CUDA device function that performs 1D stencil operation
// -------------------------------------------------------
__global__ void stencil_1D(int *in, int *out, int dim){

  //int lindex = threadIdx.x + RADIUS;
  //int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = remappingBlockIDx(blockIdx.x, BLOCKXMODE) * blockDim.x + remappingThreadIDx(threadIdx.x, THREADXMODE);
  //int stride = gridDim.x * blockDim.x;
  int result = 0;
  for (int j = -RADIUS ; j < DATA_PER_THREAD + RADIUS; j++){
  	result += in[tid * DATA_PER_THREAD + RADIUS + j];
  }
  out[tid] = result;
}

// ------------
// main program
// ------------
int main(void){

  //printf("start main\n");
  int *h_in = (int *) malloc(sizeof(int) * (N + 2 * RADIUS));
  int *h_out = (int *) malloc(sizeof(int) * (GRIDSIZE * BLOCKSIZE));
  int *d_in, *d_out;
  int size_in = (N + 2 * RADIUS) * sizeof(int);
  int size_out = (GRIDSIZE * BLOCKSIZE) * sizeof(int);
  int i, j, result, err;

  //printf("Start to alloc\n");
  hipMalloc((void **)&d_in, size_in);
  hipMalloc((void **)&d_out, size_out);

  //printf("Start to init\n");
  for (i=0; i< N + 2 * RADIUS; i++){
	h_in[i] = 1;
  }

  hipMemcpy(d_in, h_in, size_in, hipMemcpyHostToDevice);

  // Apply stencil by launching a sufficient number of blocks
  //printf("\n---------------------------\n");
  //printf("Launching 1D stencil kernel\n");
  //printf("---------------------------\n");
  //printf("Vector length	 = %d (%d MB)\n",N,N*4/1024/1024);
  //printf("Stencil radius	= %d\n",RADIUS);
  //printf("Blocks			= %d\n",GRIDSIZE);
  //printf("Threads per block = %d\n",BLOCKSIZE);
  //printf("Total threads	 = %d\n",GRIDSIZE*BLOCKSIZE);

  stencil_1D<<<GRIDSIZE,BLOCKSIZE>>>(d_in, d_out, N);

  // copy results back to host
  hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);

  // deallocate memory
  hipFree(d_in);
  hipFree(d_out);

  // check results
  err = 0;
  for (i=0; i< (GRIDSIZE * BLOCKSIZE) ; i++){
	result = 0;

	for (j = -RADIUS ; j < DATA_PER_THREAD + RADIUS; j++){
		result += h_in[i * DATA_PER_THREAD + RADIUS + j];
	}
	if (h_out[i] != result) {
	  err++;
	  printf("h_out[%d]=%d, cpu result = %d\n",i,h_out[i], result);
	}
  }

  if (err != 0){
	printf("\n Error, %d elements do not match!\n\n", err);
  } else {
	printf("\n Success! All elements match CPU result.\n\n");
  }

  return 0;

}
